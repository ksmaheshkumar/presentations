#include "hip/hip_runtime.h"
/******************************************************************************

Double salted md5 cracker by @Wireghoul - justanotherhacker.com
Based on CUDA cracker code by FireXware - OSSBox.com

******************************************************************************/

#define MAX_BRUTE_LENGTH 14
#define MAX_SALT_LENGTH 38
#define MAX_TOTAL (MAX_SALT_LENGTH + MAX_BRUTE_LENGTH + MAX_SALT_LENGTH)

//Performance:
#define BLOCKS 64
#define THREADS_PER_BLOCK 128
#define MD5_PER_KERNEL 200
#define OUTPUT_INTERVAL 20

__device__ __constant__ unsigned char cudaBrute[MAX_BRUTE_LENGTH];
__device__ __constant__ unsigned char cudaLeftSalt[MAX_SALT_LENGTH];
__device__ __constant__ unsigned char cudaRightSalt[MAX_SALT_LENGTH];
__device__ __constant__ unsigned char cudaCharSet[95];
__device__ unsigned char correctPass[MAX_TOTAL];

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "md5.cu" //This contains our MD5 helper functions
#include "md5kernel.cu" //the CUDA thread

void checkCUDAError(const char *msg);

void ZeroFill(unsigned char* toFill, int length) {
    int i = 0; 
    for (i = 0; i < length; i++)
        toFill[i] = 0;
}

bool BruteIncrement(unsigned char* brute, int setLen, int wordLength, int incrementBy) {
    int i = 0;
    while (incrementBy > 0 && i < wordLength) {
        int add = incrementBy + brute[i];
        brute[i] = add % setLen;
        incrementBy = add / setLen;
        i++;
    }
    return incrementBy != 0; //we are done, if there is a remainder, because we have looped over the max
}

int main(int argc, char** argv) {
    if (argc != 5) {
        printf("Usage: %s hash salt1 salt2 length\n", argv[0]);
        return 1;
    }
    int wordLength = atoi(argv[4]);
    int charSetLen = 0;


    int numThreads = BLOCKS * THREADS_PER_BLOCK;

    unsigned char currentBrute[MAX_BRUTE_LENGTH];
    unsigned char leftSalt[MAX_SALT_LENGTH];
    unsigned char rightSalt[MAX_SALT_LENGTH];

    unsigned char cpuCorrectPass[MAX_TOTAL];

    ZeroFill(currentBrute, MAX_BRUTE_LENGTH);
    ZeroFill(cpuCorrectPass, MAX_TOTAL);
    ZeroFill(leftSalt, MAX_SALT_LENGTH);
    ZeroFill(rightSalt, MAX_SALT_LENGTH);

    charSetLen = 82;
    unsigned char charSet[charSetLen];
    memcpy(charSet, "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ,.@!$%^&*()-_=+", charSetLen);

    unsigned char hash[32];
    //printf("%s %s %s", argv[0], argv[1], argv[2]);
    memcpy(hash, argv[1], 32);

    memcpy(leftSalt, argv[2], strlen(argv[2]));
    //memcpy(rightSalt, "|0|Test Reference|1.00|20110616221931", 37);
    memcpy(rightSalt, argv[3], strlen(argv[3]));

    printf("Cracking %s using salts %s$password$%s\n", hash, leftSalt, rightSalt);
    //turn the correct hash into it's four parts
    uint v1, v2, v3, v4;
    md5_to_ints(hash,&v1,&v2,&v3,&v4);

    //copy the salts to global
    hipMemcpyToSymbol(HIP_SYMBOL(cudaLeftSalt), &leftSalt, MAX_SALT_LENGTH, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaRightSalt), &rightSalt, MAX_SALT_LENGTH, 0, hipMemcpyHostToDevice);

    //zero the container used to hold the correct pass
    hipMemcpyToSymbol(HIP_SYMBOL(correctPass), &cpuCorrectPass, MAX_TOTAL, 0, hipMemcpyHostToDevice);

    //create and copy the charset to device
    hipMemcpyToSymbol(HIP_SYMBOL(cudaCharSet), &charSet, charSetLen, 0, hipMemcpyHostToDevice);

    bool finished = false;
    int ct = 0;
    do {
        hipMemcpyToSymbol(HIP_SYMBOL(cudaBrute), &currentBrute, MAX_BRUTE_LENGTH, 0, hipMemcpyHostToDevice);

        //run the kernel
        dim3 dimGrid(BLOCKS);
        dim3 dimBlock(THREADS_PER_BLOCK);

        crack<<<dimGrid, dimBlock>>>(numThreads, charSetLen, wordLength, v1,v2,v3,v4);

        //get the "correct pass" and see if there really is one
        hipMemcpyFromSymbol(&cpuCorrectPass, HIP_SYMBOL(correctPass), MAX_TOTAL, 0, hipMemcpyDeviceToHost);

        if (cpuCorrectPass[0] != 0) {
            printf("\n\nFOUND: ");
            int k = 0;
            while (cpuCorrectPass[k] != 0) {
                printf("%c", cpuCorrectPass[k]);
                k++;
            }
            printf("\n");
            return 0;
        }

        finished = BruteIncrement(currentBrute, charSetLen, wordLength, numThreads * MD5_PER_KERNEL);

        checkCUDAError("general");

        if (ct % OUTPUT_INTERVAL == 0) {
            printf("STATUS: %d", ct);
            //int k = 0;
            //for(k = 0; k < wordLength; k++)
            //    printf("%c",charSet[currentBrute[k]]);
            //printf("\n");
        }
        ct++;
        checkCUDAError("mehhhh");
    } while(!finished);

    return 0;
}

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}
